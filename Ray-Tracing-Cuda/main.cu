#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <memory>
#include <limits>
#include "include/vec3.cuh"
#include "include/ray.cuh"
#include "include/sphere.cuh"
#include "include/hitable.cuh"
#include "include/hitable_list.cuh"

#define RM(row,col,w) row*w+col
#define CM(row,col,h) col*h+row

void write_ppm_image(std::vector<rgb> colors, int h, int w, std::string filename) {
	std::ofstream myfile;
	myfile.open(filename + ".ppm");
	myfile << "P3\n" << w << " " << h << "\n255\n";
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			auto color = colors[RM(i, j, w)];
			myfile << color.r()*255.99 << " " << color.g()*255.99 << " " << color.b()*255.99 << std::endl;
		}
	}
	myfile.close();
}

std::vector<rgb> hello_world_render(int h, int w) {
	auto colors = std::vector<rgb>(w*h);
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			colors[RM(i, j, w)].r(j / float(w));
			colors[RM(i, j, w)].g(h - i / float(h));
			colors[RM(i, j, w)].b(0.2f);
		}
	}
	return colors;
}

rgb color(const ray& r, const std::shared_ptr<hitable>& world) {
	hit_record rec;
	if (world->hit(r, 0.0f, std::numeric_limits<float>::max(), rec)) {
		return (rec.normal + 1.0)*0.5;
	}
	vec3 unit_direction = unit_vector(r.direction());
	float t = 0.5f*(unit_direction.e[1] + 1.0f);
	return vec3(1.0f, 1.0f, 1.0f)*(1.0f - t) + vec3(0.5f, 0.7f, 1.0f)*t;
}

std::vector<rgb> simple_ray_render(int h, int w) {
	auto colors = std::vector<rgb>(w*h);
	vec3 lower_left_corner(-2.0, -1.0, -1.0);
	vec3 horizontal(4.0, 0.0, 0.0);
	vec3 vertical(0.0, 2.0, 0.0);
	vec3 origin(0.0, 0.0, 0.0);
	auto world = std::make_shared<hitable_list>();
	world->add_hitable(std::make_shared<sphere>(vec3(0, 0, -1), 0.5, rgb(1.0, 0, 0)));
	world->add_hitable(std::make_shared<sphere>(vec3(0, -100.5, -1), 100, rgb(0.0, 1.0, 0)));

	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			float u = float(j) / float(w);
			float v = float(h - i) / float(h);
			ray r(origin, lower_left_corner + (horizontal * u) + (vertical * v));
			colors[RM(i, j, w)] = color(r, world);
		}
	}
	return colors;
}

int main() {
	int h = 100;
	int w = 200;

	auto colors = simple_ray_render(h, w);
	write_ppm_image(colors, h, w, "render");
}