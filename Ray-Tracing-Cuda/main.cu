#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <memory>
#include <limits>
#include <random>
#include "include/vec3.cuh"
#include "include/ray.cuh"
#include "include/sphere.cuh"
#include "include/hitable.cuh"
#include "include/hitable_list.cuh"
#include "include/camera.cuh"

#define RM(row,col,w) row*w+col
#define CM(row,col,h) col*h+row

// std::random_device rd;	// Will be used to obtain a seed for the random number engine
std::mt19937 gen(42);		// Standard mersenne_twister_engine seeded with rd()
std::uniform_real_distribution<float> dis(0.0, 1.0);

void write_ppm_image(std::vector<rgb> colors, int h, int w, std::string filename) {
	std::ofstream myfile;
	myfile.open(filename + ".ppm");
	myfile << "P3\n" << w << " " << h << "\n255\n";
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			auto color = colors[RM(i, j, w)];
			myfile << color.r()*255.99 << " " << color.g()*255.99 << " " << color.b()*255.99 << std::endl;
		}
	}
	myfile.close();
}

std::vector<rgb> hello_world_render(int h, int w) {
	auto colors = std::vector<rgb>(w*h);
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			colors[RM(i, j, w)].r(j / float(w));
			colors[RM(i, j, w)].g(h - i / float(h));
			colors[RM(i, j, w)].b(0.2f);
		}
	}
	return colors;
}

vec3 random_in_unit_sphere() {
	vec3 p;
	do {
		p = vec3(dis(gen), dis(gen), dis(gen))*2.0 - vec3(1, 1, 1);
	} while (p.squared_length() >= 1.0);
	return p;
}

rgb color(const ray& r, const std::shared_ptr<hitable>& world) {
	hit_record rec;
	if (world->hit(r, 0.001f, std::numeric_limits<float>::max(), rec)) {
		auto target = rec.p + rec.normal + random_in_unit_sphere();
		return color(ray(rec.p, target - rec.p), world)*0.5;
	}
	vec3 unit_direction = unit_vector(r.direction());
	float t = 0.5f*(unit_direction.e[1] + 1.0f);
	return vec3(1.0f, 1.0f, 1.0f)*(1.0f - t) + vec3(0.5f, 0.7f, 1.0f)*t;
}

std::vector<rgb> simple_ray_render(int h, int w, int samples) {
	auto colors = std::vector<rgb>(w*h);
	auto c = camera();
	auto world = std::make_shared<hitable_list>();
	world->add_hitable(std::make_shared<sphere>(vec3(0, 0, -1), 0.5, rgb(1.0, 0, 0)));
	world->add_hitable(std::make_shared<sphere>(vec3(0, -100.5, -1), 100, rgb(0.0, 1.0, 0)));

	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			rgb pix(0, 0, 0);
			for (int s = 0; s < samples; s++) {
				float u = float(j + dis(gen)) / float(w);
				float v = float(h - i + dis(gen)) / float(h);
				ray r = c.get_ray(u, v);
				pix += color(r, world);
			}
			pix /= float(samples);
			pix = pix.v_sqrt(); // gamma correct (gamma 2)
			colors[RM(i, j, w)] = pix;
		}
	}
	return colors;
}

int main() {
	int h = 200;
	int w = 400;
	int s = 10;

	auto colors = simple_ray_render(h, w, s);
	write_ppm_image(colors, h, w, "render");
}